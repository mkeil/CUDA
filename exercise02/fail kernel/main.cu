#include "hip/hip_runtime.h"
__global__ void emptyKernel() {
	printf("Test");
}

int main() {
	dim3 threadsPerBlock(1);
	dim3 blocksPerGrid(1);
	emptyKernel<<<blocksPerGrid, threadsPerBlock>>>();
	hipDeviceSynchronize();
	return 0;
}