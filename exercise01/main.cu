#include "hip/hip_runtime.h"
__global__ void emptyKernel() {
}

int main() {
	dim3 threadsPerBlock(1);
	dim3 blocksPerGrid(1);
	emptyKernel<<<blocksPerGrid, threadsPerBlock>>>();
	hipDeviceSynchronize();
	return 0;
}